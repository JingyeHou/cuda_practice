#include <stdexcept>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;
bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    cout << count << endl;


    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}


int main()
{
    if(!InitCUDA()) {
        return 0;
    }

    printf("CUDA initialized.\n");

    return 0;
}
