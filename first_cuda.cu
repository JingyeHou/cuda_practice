#include "hip/hip_runtime.h"
#include <stdexcept>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define DATA_SIZE 1048576

int data[DATA_SIZE];

using namespace std;

void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    cout << count << endl;


    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

__global__ static void sumOfSquares(int *num, int* result,
    clock_t* time)
{
    int sum = 0;
    int i;
    clock_t start = clock();
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }

    *result = sum;
    *time = clock() - start;
}

int main()
{
    if(!InitCUDA()) {
        return 0;
    }

    printf("CUDA initialized.\n");

    GenerateNumbers(data, DATA_SIZE);

    int* gpudata, *result;
        clock_t* time;
        hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
        hipMalloc((void**) &result, sizeof(int));
        hipMalloc((void**) &time, sizeof(clock_t));
        hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,
            hipMemcpyHostToDevice);

        sumOfSquares<<<1, 1, 0>>>(gpudata, result, time);

        int sum;
        clock_t time_used;
        hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&time_used, time, sizeof(clock_t),
            hipMemcpyDeviceToHost);
        hipFree(gpudata);
        hipFree(result);

        printf("sum: %d time: %d\n", sum, time_used);

    return 0;
}
