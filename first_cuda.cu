#include "hip/hip_runtime.h"
//#include <stdexcept>
//#include <fstream>
//#include <iostream>
//#include <stdio.h>
//#include <hip/hip_runtime.h>
//#include <stdlib.h>
//
//#define DATA_SIZE 1048576
//
//int data[DATA_SIZE];
//
//using namespace std;
//
//
//#define NUM_THREADS 256
//
//__global__ static void matMultCUDA(const float* a, size_t lda,
//        const float* b, size_t ldb, float* c, size_t ldc, int n)
//    {
//        const int tid = threadIdx.x;
//        const int bid = blockIdx.x;
//        const int idx = bid * blockDim.x + tid;
//        const int row = idx / n;
//        const int column = idx % n;
//        int i;
//
//        if(row < n && column < n) {
//            float t = 0;
//            for(i = 0; i < n; i++) {
//                t += a[row * lda + i] * b[i * ldb + column];
//            }
//            c[row * ldc + column] = t;
//        }
//    }
//
//clock_t matmultCUDA(const float* a, int lda,
//	const float* b, int ldb, float* c, int ldc, int n)
//{
//	float *ac, *bc, *cc;
//	clock_t start, end;
//
//	start = clock();
//	hipMalloc((void**) &ac, sizeof(float) * n * n);
//	hipMalloc((void**) &bc, sizeof(float) * n * n);
//	hipMalloc((void**) &cc, sizeof(float) * n * n);
//
//	hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda,
//		sizeof(float) * n, n, hipMemcpyHostToDevice);
//	hipMemcpy2D(bc, sizeof(float) * n, b, sizeof(float) * ldb,
//		sizeof(float) * n, n, hipMemcpyHostToDevice);
//
//	int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
//	matMultCUDA<<<blocks * n, NUM_THREADS>>>
//		(ac, n, bc, n, cc, n, n);
//
//	hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n,
//	sizeof(float) * n, n, hipMemcpyDeviceToHost);
//
//	hipFree(ac);
//	hipFree(bc);
//	hipFree(cc);
//
//	end = clock();
//
//	return end - start;
//}
//
//void GenerateNumbers(int *number, int size)
//{
//    for(int i = 0; i < size; i++) {
//        number[i] = rand() % 10;
//    }
//}
//
//bool InitCUDA()
//{
//    int count;
//
//    hipGetDeviceCount(&count);
//    if(count == 0) {
//        fprintf(stderr, "There is no device.\n");
//        return false;
//    }
//
//    cout << count << endl;
//
//
//    int i;
//    for(i = 0; i < count; i++) {
//        hipDeviceProp_t prop;
//        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
//            if(prop.major >= 1) {
//                break;
//            }
//        }
//    }
//
//    if(i == count) {
//        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
//        return false;
//    }
//
//    hipSetDevice(i);
//
//    return true;
//}
//
//void matmult(const float* a, int lda, const float* b, int ldb,
//        float* c, int ldc, int n)
//    {
//        int i, j, k;
//
//        for(i = 0; i < n; i++) {
//            for(j = 0; j < n; j++) {
//                double t = 0;
//                for(k = 0; k < n; k++) {
//                    t += a[i * lda + k] * b[k * ldb + j];
//                }
//                c[i * ldc + j] = t;
//            }
//        }
//    }
//
//void matgen(float* a, int lda, int n)
//  {
//      int i, j;
//
//      for(i = 0; i < n; i++) {
//          for(j = 0; j < n; j++) {
//              a[i * lda + j] = (float) rand() / RAND_MAX +
//                  (float) rand() / (RAND_MAX * RAND_MAX);
//          }
//      }
//  }
//
//__global__ static void sumOfSquares(int *num, int* result,
//    clock_t* time)
//{
//    int sum = 0;
//    int i;
//    clock_t start = clock();
//    for(i = 0; i < DATA_SIZE; i++) {
//        sum += num[i] * num[i];
//    }
//
//    *result = 2;
//    *time = clock() - start;
//}
//
//void compare_mat(const float* a, int lda,
//       const float* b, int ldb, int n)
//   {
//       float max_err = 0;
//       float average_err = 0;
//       int i, j;
//
//       for(i = 0; i < n; i++) {
//           for(j = 0; j < n; j++) {
//               if(b[i * ldb + j] != 0) {
//                   float err = fabs((a[i * lda + j] -
//                       b[i * ldb + j]) / b[i * ldb + j]);
//                   if(max_err < err) max_err = err;
//                   average_err += err;
//               }
//           }
//       }
//
//       printf("Max error: %g Average error: %g\n",
//           max_err, average_err / (n * n));
//   }
//
//int main()
//{
//	float *a, *b, *c, *d;
//	int n = 1000;
//
//    if(!InitCUDA()) {
//        return 0;
//    }
//
//    printf("CUDA initialized.\n");
//
//    GenerateNumbers(data, DATA_SIZE);
//
//    int* gpudata, *result;
//	clock_t* time;
//	hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
//	hipMalloc((void**) &result, sizeof(int));
//	hipMalloc((void**) &time, sizeof(clock_t));
//	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,
//		hipMemcpyHostToDevice);
//
//	sumOfSquares<<<1, 1, 0>>>(gpudata, result, time);
//
//	int sum;
//	clock_t time_used;
//	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
//	hipMemcpy(&time_used, time, sizeof(clock_t),
//		hipMemcpyDeviceToHost);
//	hipFree(gpudata);
//	hipFree(result);
//
//	printf("sum: %d time: %d\n", sum, time_used);
//
//	a = (float*) malloc(sizeof(float) * n * n);
//	b = (float*) malloc(sizeof(float) * n * n);
//	c = (float*) malloc(sizeof(float) * n * n);
//	d = (float*) malloc(sizeof(float) * n * n);
//
//	srand(0);
//
//	matgen(a, n, n);
//	matgen(b, n, n);
//
//	clock_t time1 = matmultCUDA(a, n, b, n, c, n, n);
//
//	matmult(a, n, b, n, d, n, n);
//	compare_mat(c, n, d, n, n);
//
//	double sec = (double) time1 / CLOCKS_PER_SEC;
//	printf("Time used: %.2f (%.2lf GFLOPS)\n", sec,
//		2.0 * n * n * n / (sec * 1E9));
//
//
//    return 0;
//}


#include <stdexcept>
#include <fstream>
#include <iostream>
#include "utils/init_graph.hpp"
#include "utils/read_graph.hpp"
#include "utils/globals.hpp"
#include "prepare_and_process_graph.cuh"
#include "stm.cuh"

#define NUM_THREADS 256

#define DATA_SIZE 10

int data[DATA_SIZE];

//static const bool VerifyByPrim = true;
//static const bool  RunKruskalFindMSTOnly = true;

// Open files safely.
template<typename T_file>
void openFileToAccess(T_file& input_file, std::string file_name) {
	input_file.open(file_name.c_str());
	if (!input_file)
		throw std::runtime_error(
				"Failed to access specified file: " + file_name + "\n");
}

template<typename T_file>
void reOpenFileToAccess(T_file& input_file, std::string file_name) {
	input_file.close();
	openFileToAccess<std::ifstream>(input_file,file_name);
}

template<typename T_file>
void closeFile(T_file& input_file) {
	input_file.close();
}


__global__ static void sumOfSquares(int *num, int* result,
    clock_t* time)
{
    int sum = 0;
    int i;
    clock_t start = clock();
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }

    *result = 2;
    *time = clock() - start;
}

using namespace std;

__global__ static void testMark(bool* boolean, LocalVertex* localSet, Vertex* vertex, int* result) {
		int tid = threadIdx.x;
		vertex[tid]._UFDS_ParentIdx = tid - 1;
		vertex[tid].nbrVtxIdx = tid;
		vertex[tid].active = true;
		vertex[tid].lock = tid;
		vertex[tid]._CRSindex = tid;
		vertex[tid].suggestedWeight = tid;
		if (tid == 0) {
			vertex[tid]._UFDS_ParentIdx = tid;
		}
		*result = vertex[tid].nbrVtxIdx;

		uint priority = 4;
		int size = 4;
		int index = 4;
		mark(vertex, localSet, priority, size, index, *boolean);
}

__global__ static void testAddToArray(LocalVertex* localSet) {
		int size = threadIdx.x;
		uint priority = threadIdx.x;
		int index = threadIdx.x;
		int parentIndex = index - 1;
		LocalVertex local;
		local.change = 0;
		local.changed = false;
		local.priority = priority;
		local.vertexIdx = index;
		local.parentIdx = parentIndex;
		local.locked = false;
		addToArray(localSet, size, local);
}

__global__ static void testAcquireLocks(bool* boolean, LocalVertex* localSet, Vertex* vertex) {
	int size = DATA_SIZE;
		*boolean = acquireLocks(localSet, size, vertex);
}

__global__ static void testCommit(LocalVertex* localSet, Vertex* vertex) {
	int size = DATA_SIZE;
	commit(localSet, size, vertex);
}


void get(int & num){
	num = 4;
}
int main(int argc, char** argv) {

	bool* boolean;
	LocalVertex* localSet;
	Vertex * vertex;
	int* gpudata;
	bool *acquireResult;

	int num;
	bool boolResult;
	LocalVertex localSetResult[DATA_SIZE];
	bool acquireResultS;
	Vertex vertexResult[DATA_SIZE];

	hipMalloc((void**) &gpudata, sizeof(int));
	hipMalloc((void**) &vertex, sizeof(Vertex) * DATA_SIZE);
	hipMalloc((void**) &boolean, sizeof(bool));
	hipMalloc((void**) &acquireResult, sizeof(bool));
	hipMalloc((void**) &localSet, sizeof(LocalVertex) * DATA_SIZE);

//	test mark
	testMark<<<1, DATA_SIZE, 0>>>(boolean, localSet, vertex, gpudata);
	hipMemcpy(&num, gpudata, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&boolResult, boolean, sizeof(bool), hipMemcpyDeviceToHost);
	cout << boolResult << endl;

//	test addToArray
	testAddToArray<<<1, DATA_SIZE, 0>>>(localSet);
	hipMemcpy(&localSetResult, localSet, sizeof(LocalVertex) * DATA_SIZE, hipMemcpyDeviceToHost);
	cout << localSetResult[3].vertexIdx << endl;

//	test acquireLocks
	testAcquireLocks<<<1, DATA_SIZE, 0>>>(acquireResult, localSet, vertex);
	hipMemcpy(&acquireResultS, acquireResult, sizeof(bool), hipMemcpyDeviceToHost);
	cout << acquireResultS << endl;

//	test commit
	testCommit<<<1, DATA_SIZE, 0>>>(localSet, vertex);
	hipMemcpy(&vertexResult, vertex, sizeof(Vertex) * DATA_SIZE, hipMemcpyDeviceToHost);
	cout << vertexResult[0]._UFDS_ParentIdx << endl;


//	free memory
	hipFree(acquireResult);
	hipFree(vertex);
	hipFree(gpudata);
	hipFree(boolean);
	hipFree(localSet);

	return 0;
}
